/*
 * Alyxandra Spikerman
 * High Perfomance Computing
 * Homework 6 - Question 1
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 2
#define n 64

// taken from transpose.cu from HW5
#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);

#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \

#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);

// CUDA kernel
__global__ void get_a(float* a, float* b) {
    int i = blockIdx.x * TILE_SIZE + threadIdx.x;
    int j = blockIdx.y * TILE_SIZE + threadIdx.y;
    int k = blockIdx.z * TILE_SIZE + threadIdx.z;

    if (i < (n-1) && i > 0 && j < (n-1) && j > 0 && k < (n-1) && k > 0) {
        a[n*n*i + n*j + k] = 0.8 * (b[n*n*(i-1) + n*j + k] +
                                    b[n*n*(i+1) + n*j + k] +
                                    b[n*n*i + n*(j-1) + k] +
                                    b[n*n*i + n*(j+1) + k] +
                                    b[n*n*i + n*j + (k-1)] +
                                    b[n*n*i + n*j + (k+1)]);
    }
}

int main(int argc, char* argv[] ) {
    size_t total_bytes = n * n * n * sizeof(float);
    float* h_a = (float*)malloc(total_bytes);
    float* h_b = (float*)malloc(total_bytes);
    float* d_a;
    hipMalloc(&d_a, total_bytes);
    float* d_b;
    hipMalloc(&d_b, total_bytes);

    srand(150);

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < n; k++) {
                h_b[n*n*i + n*j + k] = (rand() % 50) + 1;
            }
        }
    }

    int gridSize = 1 + ((n - 1) / TILE_SIZE);
    dim3 dimGrid(gridSize, gridSize, gridSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE, TILE_SIZE);

    hipMemcpy(d_a, h_a, total_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, total_bytes, hipMemcpyHostToDevice);

    printf("\nStart kernel\n");

    float Ktime;
    TIMER_CREATE(Ktime);
    TIMER_START(Ktime);

    get_a<<< dimGrid, dimBlock >>>(d_a, d_b); // Execute the kernel
    hipDeviceSynchronize(); // wait for everything to finish before accessing

    TIMER_END(Ktime);
    printf("Kernel Execution Time: %f ms\n", Ktime);

    hipMemcpy(h_a, d_a, total_bytes, hipMemcpyDeviceToHost); // Copy histogram to host

    // for (int i = 1; i < n - 1; i++) {
    //     for (int j = 1; j < n - 1; j++) {
    //         for (int k = 1; k < n - 1; k++) {
    //             printf("%f ", h_a[n*n*i + n*j + k]);
    //         }
    //     }
    // }

    // free allocated memory
    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);
    return 0;
}
